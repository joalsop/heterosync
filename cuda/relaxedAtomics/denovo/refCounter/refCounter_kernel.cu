
#include <hip/hip_runtime.h>
#define WARP_SIZE 32
#define HALF_WARP_SIZE (WARP_SIZE >> 1)

__global__ void refCounter_kernel(int * d_counters0,
                                  int * d_counters1,
                                  int * d_del0,
                                  int * d_del1,
                                  const unsigned int numRepeats,
                                  const unsigned int numSharersPerGroup,
                                  const unsigned int numCounters,
                                  const unsigned int numSharingGroups,
                                  const unsigned int numCounters_perSharingGroup/*,
                                  const region_t counterReg,
                                  const region_t delReg*/) {
  // local variables
  const unsigned int myBaseLoc = ((blockIdx.x * blockDim.x) + threadIdx.x);
  const unsigned int mySharingGroup = (blockIdx.x % numSharingGroups);
  const unsigned int myCounterLoc = ((mySharingGroup * numCounters_perSharingGroup) + threadIdx.x);
  int * counterAddr0, * counterAddr1, * delAddr0, * delAddr1; // for PTX inlining
  __shared__ volatile int dummyLocal[256]; // for doing local dummy calculations, assumes blockDim.x <= 256

  /*
  const bool isMasterThread = (threadIdx.x == 0);

  // only use atomics and data stores in this kernel, so nothing should be
  // invalidated
  if (isMasterThread) {
    __denovo_setAcquireRegion(delReg); // written with regular stores
    __denovo_addAcquireRegion(counterReg); // written with atomics
  }
  __syncthreads();
  */
  dummyLocal[threadIdx.x] = 0;
  __syncthreads();

  // the counters each thread accesses is fixed, regardless of the number of loop iterations
  counterAddr0 = &(d_counters0[myCounterLoc]);
  counterAddr1 = &(d_counters1[myCounterLoc]);
  delAddr0 = &(d_del0[myBaseLoc]);
  delAddr1 = &(d_del1[myBaseLoc]);

  // repeat this process a few times
  for (int i = 0; i < numRepeats; ++i) {
    /*
    // use atomicAdds instead of atomicIncs because incs have store release
    // semantics -- these atomics can be reordered with each other
    atomicAdd(counterAddr0, 1);
    atomicAdd(counterAddr1, 1);
    */
    /*
      Replace the above atomics with inlined PTX to ensure that they are
      next to each other in the instruction sequence and thus can be
      overlapped.

      NOTE: Across all of the inlined assembly blocks we can't reuse the
      same temp reg names.
    */
    asm volatile(// Temp Registers
                 /*
                   t1 and t2 aren't actually used for anything (they hold
                   the results of the atomic adds, but we don't return
                   them).  Still need them for correct PTX though.
                 */
                 ".reg .u32 t1;\n\t"    // temp reg t1
                 ".reg .u32 t2;\n\t"    // temp reg t2
                 // PTX Instructions
                 "atom.add.u32 t1, [%0], 1;\n\t" // atomicAdd for counterAddr0
                 "atom.add.u32 t2, [%1], 1;"     // atomicAdd for counterAddr1
                 // no outputs
                 // inputs
                 :: "l"(counterAddr0), "l"(counterAddr1)
                 );

    // Do accesses in scratchpad here to space inc and dec out
    for (int j = 0; j < numRepeats * 2; ++j) {
      dummyLocal[threadIdx.x] += j;
      __syncthreads();
    }

    // If the shared counter == 0 (old value == 1), then mark the "object" to
    // be deleted
    /*
    // use atomicDec's with threadfences to ensure that we have acquire-release
    // semantics for DRF1 and DRF0
    unsigned int currCount0 = atomicDec(counterAddr0, 10000000000);
    unsigned int currCount1 = atomicDec(counterAddr1, 10000000000);
    if (currCount0 <= 1) {
      d_del0Local[threadIdx.x] = true; --> now uses global array in inlined due to bug with nvcc
    }

    if (currCount1 <= 1) {
      d_del1Local[threadIdx.x] = true; --> now uses global array in inlined due to bug with nvcc
    }
    */
    /*
      Replace the above atomics with inlined PTX to ensure that they are
      next to each other in the instruction sequence and thus can be
      overlapped.

      NOTE: Across all of the inlined assembly blocks we can't reuse the
      same temp reg names.
    */
    asm volatile(// Temp Registers
                 ".reg .u32 q3;\n\t"    // temp reg q3 (atomAdd(counterAddr0) result)
                 ".reg .u32 q4;\n\t"    // temp reg q4 (atomAdd(counterAddr1) result))
                 ".reg .pred p5;\n\t"   // temp predicate reg p5 (branch0)
                 ".reg .pred p6;\n\t"   // temp predicate reg p6 (branch1)
                 // PTX Instructions
                 "atom.dec.u32 q4, [%1], 10000000000;\n\t" // atomicDec for counterAddr1 (store release semantics, -1 is 10000000000)
                 "atom.dec.u32 q3, [%0], 10000000000;\n\t" // atomicDec for counterAddr0 (store release semantics, -1 is 10000000000)
                 // can't pass out two values, so need to do ifs here
                 // part1 of branch for counterAddr0 result -- set p5 to 1 if
                 // q3 > q2 (if result of atomicDec is > 1) (since q3 and q4
                 // hold the old values, we check for 1 instead of 0)
                 "setp.gt.u32 p5, q3, 1;\n\t"
                 // part2 of branch for counterAddr0 result -- don't do the
                 // scratchpad store if p5 = 1 (if result of atomicDec is > 0)
                 "@p5 bra $CounterAddr1If;\n\t"
                 // if the first atomic sub result is <= 0, then set d_del0[i]
                 // to 1 (true) -- store to global array, since the scratchpad
                 // array writes don't work properly with inlined PTX
                 "st.global.s32 [%2], 1;\n\t"
                 "$CounterAddr1If:\n\t"           // label for start of counterAddr1 if
                 // part1 of branch for counterAddr1 result -- set p6 to 1 if
                 // q4 > 1 (if result of atomicDec is > 1)  (since q3 and q4
                 // hold the old values, we check for 1 instead of 0)
                 "setp.gt.u32 p6, q4, 1;\n\t"
                 // part2 of branch for counterAddr1 result -- don't do the
                 // scratchpad store if p6 = 1 (if result of atomicDec is > 0)
                 "@p6 bra $Done;\n\t"
                 // if the second atomic sub result is <= 0, then set d_del1[1]
                 // to 1 (true) -- store to global array, since the scratchpad
                 // array writes don't work properly with inlined PTX
                 "st.global.s32 [%3], 1;\n\t"
                 "$Done:"
                 // no outputs
                 // inputs
                 :: "l"(counterAddr0), "l"(counterAddr1), "l"(delAddr0), "l"(delAddr1)
                 );
  }

  /*
  if (isMasterThread) {
    __denovo_gpuEpilogue(counterReg); // written with atomics
    __denovo_gpuEpilogue(delReg); // written with regular stores
  }
  */
}
