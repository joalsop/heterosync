#include "cudaLocks.h"

hipError_t cudaLocksInit(const int maxBlocksPerKernel, const bool pageAlign)
{
  hipError_t cudaErr = hipGetLastError();
  checkError(cudaErr, "Start cudaLocksInit");

  hipHostMalloc(&cpuLockData, sizeof(cudaLockData_t));

  if (maxBlocksPerKernel <= 0)    return hipErrorNotInitialized;

  // initialize some of the lock data's values
  cpuLockData->maxBufferSize          = maxBlocksPerKernel;
  cpuLockData->arrayStride            = (maxBlocksPerKernel + NUM_SM) / 16 * 16;

  hipMalloc(&cpuLockData->barrierBuffers,   sizeof(unsigned int) * cpuLockData->arrayStride * 2);

  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);

  cudaErr = hipGetLastError();
  checkError(cudaErr, "Before memset");

  hipDeviceSynchronize();
  hipEventRecord(start, 0);

  hipMemset(cpuLockData->barrierBuffers, 0, sizeof(unsigned int) * cpuLockData->arrayStride * 2);

  hipDeviceSynchronize();
  hipEventRecord(end, 0);
  hipEventSynchronize(end);
  float elapsedTime = 0.0f;
  hipEventElapsedTime(&elapsedTime, start, end);
  fprintf(stdout, "\tmemcpy H->D 1 elapsed time: %f ms\n", elapsedTime);
  fflush(stdout);

  hipEventDestroy(start);
  hipEventDestroy(end);

  return hipSuccess;
}

hipError_t cudaLocksDestroy()
{
  if (cpuLockData == NULL) { return hipErrorNotInitialized; }

  hipFree(cpuLockData->barrierBuffers);
  hipHostFree(cpuLockData);

  return hipSuccess;
}
